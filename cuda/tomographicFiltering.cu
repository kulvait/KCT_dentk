#include "hip/hip_runtime.h"
// Logging

#include "tomographicFiltering.cuh"

__global__ void ifftshiftSpectral(float2* __restrict__ x, const int SIZEX, const int SIZEY)
{
    const int PX = threadIdx.y + blockIdx.y * blockDim.y;
    const int PY = threadIdx.x + blockIdx.x * blockDim.x;
    const int xSizeHermitan = SIZEX / 2 + 1;
    const int IDX = xSizeHermitan * PY + PX;
    if((PX >= xSizeHermitan) || (PY >= SIZEY))
        return;
    int centerShift = (SIZEX + 1) / 2;
    int freq = PX;
    double shiftingAngle = -TWOPI * freq * centerShift / SIZEX;
    double sinVal, cosVal;
    sincos(shiftingAngle, &sinVal, &cosVal);
    float2 xval = x[IDX];
    x[IDX].x = xval.x * cosVal - xval.y * sinVal;
    x[IDX].y = xval.y * cosVal + xval.x * sinVal;
}

void CUDAifftshiftSpectral(dim3 threads, dim3 blocks, void* x, const int SIZEX, const int SIZEY)
{
    ifftshiftSpectral<<<blocks, threads>>>((float2*)x, SIZEX, SIZEY);
}

__global__ void RadonFilter(float2* __restrict__ x,
                            const int SIZEX,
                            const int SIZEY,
                            const float pixel_size_x,
                            const bool ifftshift)
{
    const int PX = threadIdx.y + blockIdx.y * blockDim.y;
    const int PY = threadIdx.x + blockIdx.x * blockDim.x;
    const int xSizeHermitan = SIZEX / 2 + 1;
    const int IDX = xSizeHermitan * PY + PX;
    if((PX >= xSizeHermitan) || (PY >= SIZEY))
        return;
    double L = ((double)SIZEX) * pixel_size_x;
    //Note that K shall be PX/L but 1/L is a global scaling factor, which is additionally used here
    //See Kak_Slaney, Ch 3, p. 66, (41)
    double K = PX / (L * L);
    //ifftshiftSpectral
    //Note normally we would call fftshift(fft(ifftshift(image))
    //Now just transfrom results as if we input sequence f[ifftshift(x)]
    if(ifftshift)
    {

        int centerShift = (SIZEX + 1) / 2;
        int freq = PX;
        double shiftingAngle = -TWOPI * freq * centerShift / SIZEX;
        double sinVal, cosVal;
        sincos(shiftingAngle, &sinVal, &cosVal);
        float2 xval = x[IDX];
        xval.x *= K;
        xval.y *= K;
        x[IDX].x = xval.x * cosVal - xval.y * sinVal;
        x[IDX].y = xval.y * cosVal + xval.x * sinVal;
    } else
    {
        x[IDX].x *= K;
        x[IDX].y *= K;
    }
}

void CUDARadonFilter(dim3 threads,
                     dim3 blocks,
                     void* x,
                     const int SIZEX,
                     const int SIZEY,
                     const float pixel_size_x,
                     bool ifftshift)
{
    /*
    printf("threads=(%d,%d,%d), blocks(%d, %d, %d) SIZEX=%d, SIZEY=%d L=%f 1/L=%f pixel_size_x=%f "
           "ifftshift=%s\n",
           threads.x, threads.y, threads.z, blocks.x, blocks.y, blocks.z, SIZEX, SIZEY,
           ((double)SIZEX) * pixel_size_x, 1 / (((double)SIZEX) * pixel_size_x), pixel_size_x,
           ifftshift ? "true" : "false");
*/
    RadonFilter<<<blocks, threads>>>((float2*)x, SIZEX, SIZEY, pixel_size_x, ifftshift);
}

inline __device__ float scaledSigmoid(float x, float scale)
{
    float minval = 1.0f / (1.0f + expf(scale));
    float sigmoid = 1.0f / (1.0f + expf(-scale * x));
    return (sigmoid - minval) / (1.0f - 2.0f * minval);
}

__global__ void ParkerFilter(
    float* __restrict__ x, const int SIZEX, const int SIZEY, const float corpos, const float zslope)
{
    const int PX = threadIdx.y + blockIdx.y * blockDim.y;
    const int PY = threadIdx.x + blockIdx.x * blockDim.x;
    const int IDX = SIZEX * PY + PX;
    float cor = corpos + PY * zslope;
    float dist;
    float radius;
    if(cor > 0.5f * (SIZEX - 1))
    {
        radius = (SIZEX - 1) - cor + 0.5;
        dist = cor - PX;
    } else
    {
        radius = cor + 0.5;
        dist = PX - cor;
    }
    if(abs(dist) <= radius)
    {
        float distFromCor = dist / radius; //[-1,1]
        float factor;
        //factor = 0.5f + 0.5f * distFromCor;//linear factor [0,1]
        factor = scaledSigmoid(distFromCor, 5.0f); //sigmoid factor [0,1]
        //	factor = 0.0f;
        x[IDX] *= factor;
    }
    /*
	else
	{
		//do nothing
		//x[IND] = x[IND];
	}
*/
}

void CUDAParkerFilter(dim3 threads,
                      dim3 blocks,
                      void* x,
                      const int SIZEX,
                      const int SIZEY,
                      const float corpos,
                      const float zslope)
{
    ParkerFilter<<<blocks, threads>>>((float*)x, SIZEX, SIZEY, corpos, zslope);
}

//circular shift
//shift .. The number of places by which elements are shifted to the right.
__global__ void roll(float* __restrict__ x_in,
                     float* __restrict__ x_out,
                     const int shift,
                     const int SIZEX,
                     const int SIZEY)
{
    const int PX = threadIdx.y + blockIdx.y * blockDim.y;
    const int PY = threadIdx.x + blockIdx.x * blockDim.x;
    if((PX >= SIZEX) || (PY >= SIZEY))
        return;
    int PX_shift;
    if(shift > PX)
    {
        PX_shift = PX + SIZEX - shift;
    } else
    {
        PX_shift = PX - shift;
    }
    int IDX_IN = SIZEX * PY + PX_shift;
    int IDX_OUT = SIZEX * PY + PX;
    x_out[IDX_OUT] = x_in[IDX_IN];
}

void CUDARoll(dim3 threads,
              dim3 blocks,
              void* x_in,
              void* x_out,
              const int shift,
              const int SIZEX,
              const int SIZEY)
{
    int modulusShift = shift;
    while(modulusShift < 0)
    {
        modulusShift += SIZEX;
    }
    while(modulusShift >= SIZEX)
    {
        modulusShift -= SIZEX;
    }
    roll<<<blocks, threads>>>((float*)x_in, (float*)x_out, modulusShift, SIZEX, SIZEY);
}

void CUDAifftshift(
    dim3 threads, dim3 blocks, void* x_in, void* x_out, const int SIZEX, const int SIZEY)
{
    int shift = (SIZEX + 1) / 2;
    roll<<<blocks, threads>>>((float*)x_in, (float*)x_out, shift, SIZEX, SIZEY);
}

void CUDAfftshift(
    dim3 threads, dim3 blocks, void* x_in, void* x_out, const int SIZEX, const int SIZEY)
{
    int shift = SIZEX / 2;
    roll<<<blocks, threads>>>((float*)x_in, (float*)x_out, shift, SIZEX, SIZEY);
}

__global__ void constantMultiplication(float* __restrict__ x,
                                       const float factor,
                                       const int SIZEX,
                                       const int SIZEY,
                                       const int TOX,
                                       const int TOY)
{
    const int PX = threadIdx.y + blockIdx.y * blockDim.y;
    const int PY = threadIdx.x + blockIdx.x * blockDim.x;
    if((PX >= TOX) || (PY >= TOY))
        return;
    const int IDX = SIZEX * PY + PX;
    x[IDX] *= factor;
}

void CUDAconstantMultiplication(dim3 threads,
                                dim3 blocks,
                                void* x,
                                const float factor,
                                const int SIZEX,
                                const int SIZEY,
                                const int TOX,
                                const int TOY)
{
    printf("CUDAconstantMultiplication threads=(%d,%d,%d), blocks(%d, %d, %d) SIZEX=%d, SIZEY=%d\n",
           threads.x, threads.y, threads.z, blocks.x, blocks.y, blocks.z, SIZEX, SIZEY);
    constantMultiplication<<<blocks, threads>>>((float*)x, factor, SIZEX, SIZEY, TOX, TOY);
}

__global__ void ZeroPad(float* __restrict__ IN,
                        float* __restrict__ OUT,
                        const int SIZEX,
                        const int SIZEXPAD,
                        const int SIZEY)
{
    const int PX = threadIdx.y + blockIdx.y * blockDim.y;
    const int PY = threadIdx.x + blockIdx.x * blockDim.x;
    if((PX >= SIZEXPAD) || (PY >= SIZEY))
        return;
    const int IDX = SIZEX * PY + PX;
    const int IDXPAD = SIZEXPAD * PY + PX;
    if(PX >= SIZEX)
    {
        OUT[IDXPAD] = 0.0f;
    } else
    {
        float val = IN[IDX];
        OUT[IDXPAD] = val;
    }
}

void CUDAZeroPad(dim3 threads,
                 dim3 blocks,
                 void* GPU_in,
                 void* GPU_out,
                 const int SIZEX,
                 const int SIZEXPAD,
                 const int SIZEY)
{
    /*
    printf("CUDAZeroPad threads=(%d,%d,%d), blocks(%d, %d, %d) SIZEX=%d, SIZEY=%d\n",
           threads.x, threads.y, threads.z, blocks.x, blocks.y, blocks.z, SIZEX, SIZEY);
*/
    ZeroPad<<<blocks, threads>>>((float*)GPU_in, (float*)GPU_out, SIZEX, SIZEXPAD, SIZEY);
}

__global__ void SymmPad(float* __restrict__ IN,
                        float* __restrict__ OUT,
                        const int SIZEX,
                        const int SIZEXPAD,
                        const int SIZEY)
{
    const int PX = threadIdx.y + blockIdx.y * blockDim.y;
    const int PY = threadIdx.x + blockIdx.x * blockDim.x;
    if((PX >= SIZEXPAD) || (PY >= SIZEY))
        return;
    const int SIZEXPERIOD = 2 * SIZEX - 2;
    const int IDXPAD = SIZEXPAD * PY + PX;
    int PX_ORIGIN = PX;
    while(PX_ORIGIN > SIZEXPERIOD)
    {
        PX_ORIGIN -= SIZEXPERIOD;
    }
    if(PX_ORIGIN >= SIZEX)
    {
        PX_ORIGIN = SIZEX - 2 - (PX_ORIGIN - SIZEX);
    }
    int IDX = SIZEX * PY + PX_ORIGIN;
    OUT[IDXPAD] = IN[IDX];
}

void CUDASymmPad(dim3 threads,
                 dim3 blocks,
                 void* GPU_in,
                 void* GPU_out,
                 const int SIZEX,
                 const int SIZEXPAD,
                 const int SIZEY)
{
    /*
    printf("CUDAZeroPad threads=(%d,%d,%d), blocks(%d, %d, %d) SIZEX=%d, SIZEY=%d\n",
           threads.x, threads.y, threads.z, blocks.x, blocks.y, blocks.z, SIZEX, SIZEY);
*/
    if(SIZEX < 2)
    {
        printf("This method is invalid for SIZEX<2 but SIZEX=%d", SIZEX);
        return;
    }
    if(SIZEXPAD != 2 * SIZEX - 2)
    {
        printf("SIZEXPAD=%d is not 2*%d-2 = %d", SIZEXPAD, SIZEX, 2 * SIZEX - 2);
    }
    SymmPad<<<blocks, threads>>>((float*)GPU_in, (float*)GPU_out, SIZEX, SIZEXPAD, SIZEY);
}

__global__ void StripPad(float* __restrict__ IN,
                         float* __restrict__ OUT,
                         const int SIZEX,
                         const int SIZEXPAD,
                         const int SIZEY)
{
    const int PX = threadIdx.y + blockIdx.y * blockDim.y;
    const int PY = threadIdx.x + blockIdx.x * blockDim.x;
    if((PX >= SIZEX) || (PY >= SIZEY))
        return;
    OUT[SIZEX * PY + PX] = IN[SIZEXPAD * PY + PX];
}

void CUDAStripPad(dim3 threads,
                  dim3 blocks,
                  void* GPU_IN,
                  void* GPU_OUT,
                  const int SIZEX,
                  const int SIZEXPAD,
                  const int SIZEY)
{
    StripPad<<<blocks, threads>>>((float*)GPU_IN, (float*)GPU_OUT, SIZEX, SIZEXPAD, SIZEY);
}

template <typename T>
__device__ T computeGaussianKernel(int u, int v, int width, int height, T sigma_x, T sigma_y)
{
    T sigma_x2 = sigma_x * sigma_x;
    T sigma_y2 = sigma_y * sigma_y;
    T val = exp(-2 * M_PI * M_PI * (sigma_x2 * u * u / width + sigma_y2 * v * v / height));
    return val;
}

//Template argument W is of the type hipfftComplex or hipfftDoubleComplex
//Assuming array is of the size SIZEX_HERMITIAN * SIZEY
template <typename T, typename W>
__global__ void
SpectralGaussianBlur2D(W* __restrict__ VEC, const int SIZEX, const int SIZEY, T sigma_x, T sigma_y)
{
    const int SIZEX_HERMITIAN = SIZEX / 2 + 1;

    const int PY = threadIdx.x + blockIdx.x * blockDim.x; // y-dimension
    const int PX = threadIdx.y + blockIdx.y * blockDim.y; // x-dimension

    if(PX >= SIZEX_HERMITIAN || PY >= SIZEY)
    {
        return;
    }

    const int IDX = SIZEX_HERMITIAN * PY + PX;

    T real_in = VEC[IDX].x;
    T imag_in = VEC[IDX].y;
    T real_ker = computeGaussianKernel<T>(PX <= SIZEX / 2 ? PX : PX - SIZEX,
                                          PY <= SIZEY / 2 ? PY : PY - SIZEY, SIZEX, SIZEY, sigma_x,
                                          sigma_y);
    VEC[IDX].x = real_in * real_ker;
    VEC[IDX].y = imag_in * real_ker;

    /*
        float imag_ker = 0.0f;  // Imaginary part of Gaussian kernel is zero
        OUT[IDX].x = real_in * real_ker - imag_in * imag_ker;
        OUT[IDX].y = real_in * imag_ker + imag_in * real_ker;
*/
}

template <typename T, typename W>
void CUDASpectralGaussianBlur2D(
    dim3 threads, void* GPU_vec, const int SIZEX, const int SIZEY, const T sigma_x, const T sigma_y)
{
    printf("CUDASpectralGaussianBlur threads=(%d, %d, %d), SIZEX=%d, SIZEY=%d, sigma_x=%f, "
           "sigma_y=%f\n",
           threads.x, threads.y, threads.z, SIZEX, SIZEY, sigma_x, sigma_y);

    // Calculate the number of blocks needed
    const int SIZEX_HERMITIAN = SIZEX / 2 + 1;
    dim3 numBlocks((SIZEY + threads.x - 1) / threads.x,
                   (SIZEX_HERMITIAN + threads.y - 1) / threads.y);

    SpectralGaussianBlur2D<T, W>
        <<<numBlocks, threads>>>((T*)GPU_vec, SIZEX, SIZEY, sigma_x, sigma_y);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch SpectralGaussianBlur kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

